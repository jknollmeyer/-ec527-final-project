#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <iostream>
#include <queue>
#include <vector>
#include <climits>
#include <time.h>
#include <boost/random/uniform_real.hpp>
#include <boost/random/variate_generator.hpp>
#include <boost/random/uniform_int.hpp>
#include <boost/random/linear_congruential.hpp>

#include "dijkstra_pthread.cpp"

#define GIG 1000000000
#define CPG 3.07           // Cycles per GHz -- Adjust to your computer

#define NUM_THREADS_PER_BLOCK 	1000
#define NUM_BLOCKS 				1
#define PRINT_TIME 				1
#define SM_ARR_LEN				1000


#define numNodes 6          
#define edgeDensity 0.5     
#define maxEdgeWeight 20    

#define INF INT_MAX //Infinity
 
const int sz=10001; //Maximum possible number of vertices. Preallocating space for DataStructures accordingly


using namespace boost;

typedef boost::minstd_rand base_generator_type;


//Custom Comparator for Determining priority for priority queue (shortest edge comes first)
class prioritize {
    public: 
        bool operator ()(pair<int, int>&p1 ,pair<int, int>&p2){
            return p1.second>p2.second;
        }
};

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define IMUL(a, b) __mul24(a, b)

struct timespec diff(struct timespec start, struct timespec end);
int * dijkstra(int source, int n, vector<pair<int,int> > a[],int dis[]);

__global__ void kernel_dijkstra (float* d_A, float* d_B, float* d_C, int width) {
  	float sum = 0;
  	long int k;
  	int row = blockIdx.y*(width/2)+threadIdx.y;
  	int col = blockIdx.x*(width/2)+threadIdx.x;
  	for (k = 0; k < width; k++){
		float da_element = d_A[row*width+k];
		float db_element = d_B[k*width+col];
		sum+=da_element	* db_element;
  	}
  	d_C[row*width+col] = sum;
}


int * main(int source, int n, vector<pair<int,int> > a[],int dis[]){
	int arrLen = 0;
		
	// GPU Timing variables
	hipEvent_t startFull, stopFull, startMMM, stopMMM;
	float elapsed_gpuFull, elapsed_gpuMMM;
	
	// Arrays on GPU global memoryc
	float *d_v;
	float *d_y;
	float *d_result;

	// Arrays on the host memory
	float *h_v;
	float *h_y;
	float *h_result;
	float *h_result_gold;

	int i, errCount = 0, zeroCount = 0;
	
	if (argc > 1) {
		arrLen  = atoi(argv[1]);
	}
	else {
		arrLen = SM_ARR_LEN;
	}

	printf("Length of the array = %d\n", arrLen);

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

	// Allocate GPU memory
	size_t allocSize_float = arrLen * arrLen*sizeof(float);
	size_t allocSize_int = sizeof(int);
	CUDA_SAFE_CALL(hipMalloc((void **)&d_v, allocSize_float));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, allocSize_float));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_result, allocSize_float));
		
	// Allocate arrays on host memory
	h_v                          = (float *) malloc(allocSize_float);
	h_y                        = (float *) malloc(allocSize_float);
	h_result                   = (float *) malloc(allocSize_float);
	h_result_gold              = (float *) malloc(allocSize_float);
	
	// Initialize the host arrays
	printf("\nInitializing the arrays ...");
	// Arrays are initialized with a known seed for reproducability
	initializeArray2D(h_v, arrLen, 2453);
	initializeArray2D(h_y, arrLen, 1467);
	printf("\t... done\n\n");
	
	
#if PRINT_TIME
	// Create the cuda events
	hipEventCreate(&startFull);
	hipEventCreate(&stopFull);
	hipEventCreate(&startMMM);
	hipEventCreate(&stopMMM);
	// Record event on the default stream
	hipEventRecord(startFull, 0);
#endif
	
	// Transfer the arrays to the GPU memory
	CUDA_SAFE_CALL(hipMemcpy(d_v, h_v, allocSize_float, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, allocSize_float, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpy(d_result, h_result, allocSize_float, hipMemcpyHostToDevice));
	hipEventRecord(startMMM, 0);
	// Launch the kernel
	kernel_mmm<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_v,d_y,d_result,arrLen);
	hipEventRecord(stopMMM,0);
	hipEventSynchronize(stopMMM);
	// Check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());
	
	// Transfer the results back to the host
	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, allocSize_float, hipMemcpyDeviceToHost));

#if PRINT_TIME
	// Stop and destroy the timer
	hipEventRecord(stopFull,0);
	hipEventSynchronize(stopFull);
	hipEventElapsedTime(&elapsed_gpuFull, startFull, stopFull);
	hipEventElapsedTime(&elapsed_gpuMMM, startMMM, stopMMM);
	printf("\nGPU total time: %f (msec)\n", elapsed_gpuFull);
	printf("\nGPU MMM time: %f (msec)\n", elapsed_gpuMMM);
	hipEventDestroy(startFull);
	hipEventDestroy(stopFull);
	hipEventDestroy(startMMM);
	hipEventDestroy(stopMMM);
#endif

	struct timespec time1, time2;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
    mmm(h_v,h_y,h_result_gold,arrLen);
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
    struct timespec time_stamp = diff(time1,time2);
    printf("CPU time: %ld\n", (long int)((double)(CPG)*(double)
		 (GIG * time_stamp.tv_sec + time_stamp.tv_nsec)));

	// Compare the results
	for(i = 0; i < arrLen; i++) {
		if (abs(h_result_gold[i] - h_result[i]) > TOL) {
			errCount++;
		}
		if (h_result[i] == 0) {
			zeroCount++;
		}
	}
	
	
	for(i = 0; i < 10; i++) {
		printf("%d:\t%.8f\t%.8f\n", i, h_result_gold[i], h_result[i]);
	}
	
	
	if (errCount > 0 || zeroCount > 0) {
		if(errCount > 0 )
			printf("\n@ERROR: TEST FAILED: %d results did not matched\n", errCount);
		if (zeroCount > 0)
			printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
	}
	else {
		printf("\nTEST PASSED: All results matched\n");
	}
	//printf("\n iters = %d\n", h_iters);
	// Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(d_v));
	CUDA_SAFE_CALL(hipFree(d_y));
	CUDA_SAFE_CALL(hipFree(d_result));
		   
	free(h_v);
	free(h_y);
	free(h_result);
	free(h_result_gold);

	return 0;
}

int * Dijkstra(int source, int n, vector<pair<int,int> > a[],int dis[]) //Algorithm for SSSP
{
    bool vis[sz] = {0};
    for(int i=0;i<sz;i++) //Set initial distances to Infinity
        dis[i]=INF;
    priority_queue<pair<int,int> ,vector<pair<int,int> >, prioritize> pq; //Priority queue to store vertex,weight pairs
    pq.push(make_pair(source,dis[source]=0)); //Pushing the source with distance from itself as 0
    while(!pq.empty())
    {
        pair<int, int> curr=pq.top(); //Current vertex. The shortest distance for this has been found
        pq.pop();
        int cv=curr.first,cw=curr.second; //'cw' the final shortest distance for this vertex
        if(vis[cv]) //If the vertex is already visited, no point in exploring adjacent vertices
            continue;
        vis[cv]=true;
        for(int i=0;i<a[cv].size();i++) //Iterating through all adjacent vertices
            if(!vis[a[cv][i].first] && a[cv][i].second+cw<dis[a[cv][i].first]) //If this node is not visited and the current parent node distance+distance from there to this node is shorted than the initial distace set to this node, update it
                pq.push(make_pair(a[cv][i].first,(dis[a[cv][i].first]=a[cv][i].second+cw))); //Set the new distance and add to priority queue
    }
    return dis;
}

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}